#include "hip/hip_runtime.h"

#include "common.h"
#include ""

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void) {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

	if (!prop.deviceOverlap)
		printf("Device will not handle overlaps, so no speed up from streams\n");

	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	hipStream_t stream;
	HANDLE_ERROR(hipStreamCreate(&stream));

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

	for (int i = 0; i<FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	HANDLE_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < FULL_DATA_SIZE; i += N) {
		HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
		HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

		kernel << <N / 256, 256, 0, stream >> >(dev_a, dev_b, dev_c);

		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
	}
	HANDLE_ERROR(hipStreamSynchronize(stream));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time taken:  %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));

	HANDLE_ERROR(hipStreamDestroy(stream));

	return 0;
}
