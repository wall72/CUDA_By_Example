#include "hip/hip_runtime.h"

#include "common.h"
#include ""

#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < size) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

float malloc_test(int size) {
	hipEvent_t start, stop;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	a = (float *)malloc(N * sizeof(float));
	b = (float *)malloc(N * sizeof(float));
	partial_c = (float *)malloc(blocksPerGrid * sizeof(float));
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void **)&dev_partial_c, blocksPerGrid * sizeof(float)));

	HANDLE_ERROR(hipEventRecord(start, 0));

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

	dot << <blocksPerGrid, threadsPerBlock >> >(size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}
	printf("Value calculated:  %f\n", c);

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_partial_c));

	free(a);
	free(b);
	free(partial_c);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	return elapsedTime;
}

float host_alloc_test(int size) {
	hipEvent_t start, stop;
	float elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	HANDLE_ERROR(hipHostAlloc((void **)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void **)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void **)&partial_c, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

	HANDLE_ERROR(hipEventRecord(start, 0));

	dot << <blocksPerGrid, threadsPerBlock >> >(size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}
	printf("Value calculated:  %f\n", c);

	HANDLE_ERROR(hipHostFree(a));
	HANDLE_ERROR(hipHostFree(b));
	HANDLE_ERROR(hipHostFree(partial_c));

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	return elapsedTime;
}

int main(void) {
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

	if (prop.integrated == 1)
		printf("Integrated device\n");
	else
		printf("Discrete device\n");

	if (prop.canMapHostMemory != 1) {
		printf("Device can not map memory.\n");
		return 0;
	}

	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

	float elapsedTime;

	elapsedTime = malloc_test(N);
	printf("Time using hipMalloc: %3.1f ms\n", elapsedTime);

	elapsedTime = host_alloc_test(N);
	printf("Time using hipHostAlloc: %3.1f ms\n", elapsedTime);

	return 0;
}