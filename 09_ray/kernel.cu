#include "hip/hip_runtime.h"

#include "common.h"
#include ""
#include "cpu_bitmap.h"

#define DIM 1024

#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
	float x, y, z;
	float radius;
	float r, b, g;

	__device__ float hit(float ox, float oy, float *n) {
		float dx = ox - x;
		float dy = oy - y;

		if (dx * dx + dy * dy < radius * radius) {
			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}

		return -INF;
	}
};

#define SPHERES 20
__constant__ Sphere s[SPHERES];

__global__ void kernel(unsigned char *ptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	float r = 0, g = 0, b = 0;
	float maxz = -INF;

	for (int i = 0; i < SPHERES; i++) {
		float n;
		float t = s[i].hit(ox, oy, &n);

		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}

int main(void) {
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));

	Sphere *temp_s = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i<SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(1000.0f) - 500;
		temp_s[i].y = rnd(1000.0f) - 500;
		temp_s[i].z = rnd(1000.0f) - 500;
		temp_s[i].radius = rnd(100.0f) + 20;
	}
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere) * SPHERES));
	free(temp_s);

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel << <grids, threads >> >(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time to generate:  %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	HANDLE_ERROR(hipFree(dev_bitmap));

	bitmap.display_and_exit();
}